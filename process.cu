#include "hip/hip_runtime.h"
// -----------------------------------------------------------------
// process - A billion rows challenge related processor
// Copyright (C) 2024 Gabriele Bonacini
//
// This program is free software; you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation; either version 3 of the License, or
// (at your option) any later version.
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// You should have received a copy of the GNU General Public License
// along with this program; if not, write to the Free Software Foundation,
// Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301  USA
// -----------------------------------------------------------------

#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <linux/mman.h>

#include <cstdint>
#include <cstdlib>
#include <string>
#include <iostream>
#include <fstream>
#include <filesystem>
#include <unordered_map>
#include <unordered_set>
#include <vector>
#include <algorithm>
#include <utility>
#include <thread>

using std::cerr,
      std::string, 
      std::unordered_map, 
      std::vector, 
      std::unordered_set, 
      std::sort,
      std::pair,
      std::get,
      std::stoi,
      std::filesystem::is_regular_file,
      std::thread,
      std::ofstream,
      std::chrono::high_resolution_clock,
      std::chrono::duration_cast,
      std::chrono::duration,
      std::chrono::milliseconds;

__global__ void process(const uint16_t*   stats, const int32_t*  tempers, size_t rows, 
                        int32_t*          max,   int32_t*        min,     unsigned long long* sum,
                        uint32_t*         count){

   unsigned int cidx { blockIdx.x * blockDim.x + threadIdx.x },
                cblk { blockDim.x * gridDim.x };

   for(size_t idx{cidx}; idx<rows; idx += cblk){
      size_t outIdx = stats[idx];
      atomicMax(max + outIdx, tempers[idx]);
      atomicMin(min + outIdx, tempers[idx]);
      atomicAdd(count + outIdx, 1);
      atomicAdd(sum + outIdx, (unsigned long long) tempers[idx]);
   }
}

int main(int argc, char** argv){

   auto printError { [&](const char* msg){ cerr << msg << '\n' << "Syntax:\n" << argv[0] << " <filename>\n"; 
                                           exit(1); 
                                         } 
                   };

   if(argc != 2)                     printError("Invalid argument list");
   if(! is_regular_file(argv[1]))    printError("Invalid file path");

   const size_t       LINES          { 1'000'000'000 };

   uint16_t           *stations      { nullptr };

   int32_t            *temperat      { nullptr },
                      *max           { nullptr },
                      *min           { nullptr }; 
   uint32_t           *count         { nullptr };

   unsigned long long *sum           { nullptr };

   int ifile                         { open(argv[1], O_RDONLY | O_LARGEFILE ) };
   if( ifile == -1){
        cerr << "Error: opening input file.\n";
        exit(EXIT_FAILURE);
   }

   posix_fadvise(ifile, 0, 0, POSIX_FADV_SEQUENTIAL);
   using Stat=struct stat;
   Stat           istat;
   fstat(ifile, &istat);
   ssize_t         isize  { istat.st_size };            

   unsigned char* idata  { static_cast<unsigned char*>(mmap(nullptr, isize, PROT_READ, MAP_PRIVATE |  MAP_POPULATE, ifile, 0)) };
   if( idata == MAP_FAILED){
        cerr << "Error: mmap : " << strerror(errno) << '\n';
        exit(EXIT_FAILURE);
   }

   if( hipMallocManaged(&stations, LINES * sizeof(uint16_t)) != hipSuccess){
        cerr << "Error: allocating unified memory  (stations)\n";
        exit(EXIT_FAILURE);
   }

   if( hipMallocManaged(&temperat, LINES * sizeof(int32_t)) != hipSuccess){
        cerr << "Error: allocating unified memory  (temperat)\n";
        exit(EXIT_FAILURE);
   }

   const size_t                   THREADS  { 32 };
   vector<vector<string>>         cities(THREADS);
   vector<unordered_set<string>>  singleCities(THREADS);
   vector<vector<int32_t>>        values(THREADS);

   
   vector<pair<size_t, size_t>>   iOffsets(THREADS);
   vector<pair<size_t, size_t>>   oOffsets(THREADS);
   const size_t                   START   { 0 },
                                  STOP    { 1 },
                                  SLICE   { isize / THREADS },
                                  DELTA   { 128 };

   cerr << "Start Slices calc.\n";
   for(size_t thr{ 0 }, prev{ 0 }, slice { SLICE }; thr < THREADS; thr++, slice+=SLICE){
        get<START>(iOffsets[thr])   = prev;
        size_t sl { slice };
        for( ; idata[sl] != '\n' && sl < isize ; sl++ ) {}
        get<STOP>(iOffsets[thr])    =  sl;
        prev                        =  sl + 1;
   }
   get<STOP>(iOffsets[THREADS - 1]) =  isize - 1;
   
   cerr << "End Slices calc.\n";
  
   const size_t                  BUFF_SIZE { 128 };
   auto worker { [&](size_t thrnum, size_t begin, size_t end) { 
                      string buff,
                             kkey;
                             
                      buff.reserve(BUFF_SIZE);
                      kkey.reserve(BUFF_SIZE);

                      cities[thrnum].reserve(SLICE + DELTA);
                      values[thrnum].reserve(SLICE + DELTA);

                      for(size_t idx { begin } ; idx < end ; idx++){
                            switch(idata[idx]){
                                case ';': 
                                       kkey = buff;
                                       buff.clear();
                                break;
                                case '\n': 
                                       buff.erase( buff.size() - 2, 1);
                                       cities[thrnum].push_back(kkey);
                                       singleCities[thrnum].insert(kkey);
                                       values[thrnum].push_back(stoi(buff));
                                       buff.clear();
                                break;
                                default: 
                                   buff.push_back(idata[idx]);
                            }
                      }
                 } 
   };

   vector<thread*>   workers(THREADS);
   for(size_t thr{ 0 }; thr < THREADS; thr++)
       workers[thr] = new thread(worker, thr, get<START>(iOffsets[thr]), get<STOP>(iOffsets[thr]));
   cerr << "Threads start.\n";

   for(size_t thr{ 0 }; thr < THREADS; thr++){
       workers[thr]->join();
       delete workers[thr];
   }
   cerr << "Threads end.\n";
 
   unordered_map<string, size_t> lookup;
   vector<string>                orderedLookup;
   unordered_set<string>         singleCitiesUnited;

   for(size_t idx{0}; idx < THREADS ; idx++){
      for(auto& elem: singleCities[idx])
         singleCitiesUnited.insert(elem);
   }

   cerr << "Start Output Offsets  calc.\n";
   for(size_t thr{ 0 }, prev{ 0 }; thr < THREADS; thr++){
        get<START>(oOffsets[thr])   =  prev;
        get<STOP>(oOffsets[thr])    =  prev + cities[thr].size() - 1;
        prev                        += cities[thr].size();
   }
   get<STOP>(oOffsets[THREADS -1])  =  LINES;
   cerr << "End Output Offsets  calc.\n";

   size_t     ord            { 0 };
   for(auto& city: singleCitiesUnited){
       if(! lookup.contains(city)){
            lookup[city] = ord;
            ord++;
            orderedLookup.push_back(city);
       }
   }

   auto c1 { high_resolution_clock::now() };
   auto loadVals { [&](size_t thrnum) { 
                             for(size_t idx { get<START>(oOffsets[thrnum]) }, didx { 0 } ; idx <= get<STOP>(oOffsets[thrnum]) ; idx++, didx++)
                                    *(temperat + idx ) = values[thrnum][didx];
                     }
   };
   cerr << "Loading sensors end.\n";

   auto loadCities { [&](size_t thrnum) { 
                             for(size_t idx { get<START>(oOffsets[thrnum]) }, didx { 0 } ; idx <= get<STOP>(oOffsets[thrnum]) ; idx++, didx++)
                                    *(stations + idx ) = lookup[cities[thrnum][didx]];
                        }
   };
   cerr << "Loading cities end.\n";

   for(size_t thr{ 0 }; thr < THREADS; thr++)
       workers[thr] = new thread(loadVals, thr );
   cerr << "Threads start.\n";

   for(size_t thr{ 0 }; thr < THREADS; thr++){
       workers[thr]->join();
       delete workers[thr];
   }

   for(size_t thr{ 0 }; thr < THREADS; thr++)
       workers[thr] = new thread(loadCities, thr );
   cerr << "Threads start.\n";

   for(size_t thr{ 0 }; thr < THREADS; thr++){
       workers[thr]->join();
       delete workers[thr];
   }
   cerr << "Threads end.\n";

   auto c2 { high_resolution_clock::now() };
   auto cms_int { duration_cast<milliseconds>(c2 - c1) };
   cerr << "\nCities loading  Execution Time: " << cms_int.count() << "ms\n\n";
        
   sort(orderedLookup.begin(), orderedLookup.end());

   const size_t OUTPUTSIZE { lookup.size() };
   if( hipMallocManaged(&max, OUTPUTSIZE * sizeof(int32_t)) != hipSuccess){
        cerr << "Error: allocating unified memory  (max)\n";
        exit(EXIT_FAILURE);
   }

   if( hipMallocManaged(&min, OUTPUTSIZE * sizeof(int32_t)) != hipSuccess){
        cerr << "Error: allocating unified memory  (min)\n";
        exit(EXIT_FAILURE);
   }

   if( hipMallocManaged(&sum, OUTPUTSIZE * sizeof(unsigned long long)) != hipSuccess){
        cerr << "Error: allocating unified memory  (sum)\n";
        exit(EXIT_FAILURE);
   }

   if( hipMallocManaged(&count, OUTPUTSIZE * sizeof(uint32_t)) != hipSuccess){
        cerr << "Error: allocating unified memory  (count)\n";
        exit(EXIT_FAILURE);
   }

   const size_t BLOCKS { 256 },
                DIM    { (LINES + BLOCKS - 1) / BLOCKS };

   auto k1 { high_resolution_clock::now() };

   process<<<DIM, BLOCKS>>>(stations, temperat, LINES, max, min, sum, count);
   hipDeviceSynchronize();

   auto k2 { high_resolution_clock::now() };
   auto kms_int { duration_cast<milliseconds>(k2 - k1) };
   cerr << "\nKernel Execution Time: " << kms_int.count() << "ms\n\n";

   string ofile{argv[1]};
   ofile.append(".out");
   ofstream output(ofile);
   for(auto& city : orderedLookup)
       output << city << ';' << min[lookup[city]] / 10.0 << ';' 
              << (long long)sum[lookup[city]] / count[lookup[city]] / 10.0 << ';' 
              << max[lookup[city]] / 10.0 << '\n';

   output.close();
   close(ifile);

   hipFree(stations);
   hipFree(temperat);
   hipFree(max);
   hipFree(min);
   hipFree(sum);
   hipFree(count);

   return EXIT_SUCCESS;
}
